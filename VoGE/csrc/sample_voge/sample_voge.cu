#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <sstream>
#include <tuple>


__device__ void inline dotvectoratom(
    const float val,
    const float* source,
    float* target,
    int32_t size
){
    for (int i = 0; i < size; ++i){
        atomicAdd(target + i, source[i] * val);
    }
}

__global__ void SampleVogeKernel(
    const float* image, 
    const float* vert_weight,
    const int32_t* vert_index, 
    const int N,
    const int C,
    const int H,
    const int W,
    const int K,
    float* vert_feature,  // (L, C)
    float* vert_weight_sum // (L)
){
    const int num_threads = gridDim.x * blockDim.x;
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;

    for (int pid = tid; pid < H * W * K; pid += num_threads) {
        const int n = pid / (H * W * K);
        const int yi = (pid % (H * W * K)) / (W * K);
        const int xi = (pid % (W * K)) / K;

        const int idx_point = vert_index[pid];
        const float weight_ = vert_weight[pid];
        const int pixel_idx = n * H * W + yi * W + xi;

        if (idx_point == -1){
            continue;
        }
        dotvectoratom(weight_, image + (pixel_idx * C), vert_feature + (idx_point * C), C);

        atomicAdd(vert_weight_sum + idx_point, weight_);
    }
}


std::tuple<at::Tensor, at::Tensor> SampleVoge(
    const at::Tensor& image, // (N, W, H, C)
    const at::Tensor& vert_weight, // (N, W, H, K)
    const at::Tensor& vert_index,  // (N, W, H, K)
    const int num_vert
){
    at::cuda::CUDAGuard device_guard(image.device());
    hipStream_t stream = at::cuda::getCurrentCUDAStream();

    const int N = image.size(0);
    const int H = image.size(1);
    const int W = image.size(2);
    const int C = image.size(3);
    const int K = vert_weight.size(3);
    const int L = num_vert;

    auto float_opts = vert_weight.options().dtype(at::kFloat);

    at::Tensor vert_feature = at::zeros({L, C}, float_opts);
    at::Tensor vert_weight_sum = at::zeros({L}, float_opts);

    const size_t blocks = 1024;
    const size_t threads = 64;

    SampleVogeKernel<<<blocks, threads, 0, stream>>>(
        image.contiguous().data_ptr<float>(),
        vert_weight.contiguous().data_ptr<float>(),
        vert_index.contiguous().data_ptr<int32_t>(),
        N,
        C,
        H,
        W,
        K,
        vert_feature.contiguous().data_ptr<float>(),
        vert_weight_sum.contiguous().data_ptr<float>()
    );
    AT_CUDA_CHECK(hipGetLastError());
    return std::make_tuple(vert_feature, vert_weight_sum);
}


__global__ void SampleVogeBackwardKernel(
    const float* image, 
    const float* vert_weight,
    const int32_t* vert_index, 
    const int C,
    const int N,
    const int H,
    const int W,
    const int K,
    const float* grad_feature, 
    const float* grad_weight_sum, 
    float* grad_image,
    float* grad_vert_weight
){
    const int num_threads = gridDim.x * blockDim.x;
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;

    for (int pid = tid; pid < H * W * K; pid += num_threads) {
        const int n = pid / (H * W * K);
        const int yi = (pid % (H * W * K)) / (W * K);
        const int xi = (pid % (W * K)) / K;
        const int k = pid % K;

        const int idx_point = vert_index[pid];
        const float weight_ = vert_weight[pid];
        const int pixel_idx = n * H * W + yi * W + xi;

        if (idx_point == -1){
            continue;
        }
        dotvectoratom(weight_, grad_feature + idx_point * C, grad_image + pixel_idx * C, C);
        
        float sum_grad = grad_weight_sum[idx_point];
        for(int c =0; c < C; ++c){
            sum_grad += grad_feature[idx_point * C + c] * image[pixel_idx * C + c];
        }
        atomicAdd(grad_vert_weight + pid, sum_grad);
    }
}


std::tuple<at::Tensor, at::Tensor> SampleVogeBackward(
    const at::Tensor& image, // (N, W, H, C)
    const at::Tensor& vert_weight, // (N, W, H, K)
    const at::Tensor& vert_index,  // (N, W, H, K)
    const at::Tensor& grad_feature, // (L, C)
    const at::Tensor& grad_weight_sum // (L, )
){
    at::cuda::CUDAGuard device_guard(image.device());
    hipStream_t stream = at::cuda::getCurrentCUDAStream();

    const int N = image.size(0);
    const int H = image.size(1);
    const int W = image.size(2);
    const int C = image.size(3);
    const int K = vert_weight.size(3);
    
    auto float_opts = vert_weight.options().dtype(at::kFloat);

    at::Tensor grad_image = at::zeros({N, H, W, C}, float_opts);
    at::Tensor grad_vert_weight = at::zeros({N, H, W, K}, float_opts);

    const size_t blocks = 1024;
    const size_t threads = 64;

    SampleVogeBackwardKernel<<<blocks, threads, 0, stream>>>(
        image.contiguous().data_ptr<float>(),
        vert_weight.contiguous().data_ptr<float>(),
        vert_index.contiguous().data_ptr<int32_t>(),
        C,
        N,
        H,
        W,
        K,
        grad_feature.contiguous().data_ptr<float>(),
        grad_weight_sum.contiguous().data_ptr<float>(),
        grad_image.data_ptr<float>(),
        grad_vert_weight.data_ptr<float>()
    );
    AT_CUDA_CHECK(hipGetLastError());
    return std::make_tuple(grad_image, grad_vert_weight);
}
